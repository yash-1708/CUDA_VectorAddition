#include "hip/hip_runtime.h"

#include <hip/hip_runtime.h>
#include <>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <ctime>
#define N 100000000//size of vectors declared globally
#define M 1024//threads per block

//main kernel that runs on GPU
__global__ void VecAdd(int* a, int* b, int* c) {
	int i = (blockIdx.x * blockDim.x) + threadIdx.x;
	c[i] = a[i] + b[i];
	//printf("\n block: %d \t a:%d \t + \t b:%d \t = \t c:%d", blockIdx.x, a[i], b[i], c[i]);
}

void random_ints(int* arr, int size)
{
	int i;
	for (i = 0; i < size; ++i)
		arr[i] = rand() % 100;
}

int main() {
	clock_t start, stop;

	int *a_CPU, *b_CPU, *c_CPU, *d_CPU; //CPU vectors pointers
	//memory allocation in CPU
	a_CPU = (int*)malloc(N * sizeof(int));
	b_CPU = (int*)malloc(N * sizeof(int));
	c_CPU = (int*)malloc(N * sizeof(int));
	d_CPU = (int*)malloc(N * sizeof(int));

	int *a_GPU, *b_GPU, *c_GPU; //pointers to be stored in GPU
	//memory allocation in GPU
	hipMalloc((void**)&a_GPU, N * sizeof(int));
	hipMalloc((void**)&b_GPU, N * sizeof(int));
	hipMalloc((void**)&c_GPU, N * sizeof(int));

	//vector data allocation
	random_ints(a_CPU, N);
	random_ints(b_CPU, N);
	
	//copy CPU arry references into GPU pointers
	hipMemcpy(a_GPU, a_CPU, N * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(b_GPU, b_CPU, N * sizeof(int), hipMemcpyHostToDevice);

	//kernel call
	start = std::clock();
	VecAdd<<<(N+M-1)/M,M>>>(a_GPU, b_GPU, c_GPU);
	hipDeviceSynchronize();
	stop = std::clock();
	long float timeP = stop - start;

	//copy result from GPU to CPU
	hipMemcpy(c_CPU, c_GPU, N * sizeof(int), hipMemcpyDeviceToHost);

	//unparallel operation
	start = std::clock();
	for (size_t i = 0; i < N; i++)
	{
		d_CPU[i] = a_CPU[i] + b_CPU[i];
	}
	stop = std::clock();
	long float timeN = stop - start;
	
	//getting GPU properties and storing in prop
	hipDeviceProp_t prop;
	hipGetDeviceProperties(&prop, 0);
	int cores = prop.multiProcessorCount * 128;
	float totalCost = cores * timeP;

	//results printing
	printf("\n********************************************************************************************************\n");
	printf("N \t\t\t Nor Time \t Par Time \t Cores \t\t Tot Cost \t Speedup \t Efficiency \n");
	printf("%-20d \t %-7.3f \t %-7.3f \t %-10d \t %-7.3f \t %-7.3f \t %-5.5f \n", N, timeN, timeP, cores, totalCost, timeN / timeP, timeN / (timeP * cores));
	printf("\n********************************************************************************************************\n");

	//free memory
	hipFree(a_GPU);
	hipFree(b_GPU);
	hipFree(c_GPU);
	free(a_CPU);
	free(b_CPU);
	free(c_CPU);
	free(d_CPU);

	return 0;
}